#include "hip/hip_runtime.h"
// #include "math_functions.h"
#include "hiprand/hiprand_kernel.h"
// #include "hip/hip_vector_types.h"
//#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// universal gravitational constant in km
#define G  6.67408E-20
#define Epsilon 47.0975
#define D 376.78
#define Msi 7.4161E19   
#define Mfe 1.9549E20
#define Ksi 2.9114E14
#define Kfe 5.8228E14
#define KRPsi 0.01
#define KRPfe 0.02
#define SDPsi 0.001
#define SDPfe 0.002
#define Time_step 5.8117
#define R 3185.5 
#define R1 3185.5 
#define R2 6371.0 
#define PI 3.14
#define Init_V 3.2416
#define CENTER_MASS_X 2392.5
#define CENTER_MASS_Z 9042.7
#define OMEGA 1 
#define P_NUM 10000
#define TPB 256
#define softeningSquared 0.01f


//6.67408E-20 * 1.9549E20/
//accel = G * M / sqr_r - 0.5*(K1+K2)*(D*D-sqr_r) 

struct vecfloat3
{
	float x;
	float y;
	float z;
};

struct Particle
{
  struct vecfloat3 position;
  struct vecfloat3 velocity;
  bool p_type;   // true :silicate  false :iron
};


// particles' position & velocity
__device__ void generate_uniform_random_number(unsigned seed, int i, double* rho1, double* rho2, double* rho3 ){
    hiprandState state;
    hiprand_init(seed, i, 0, &state);
    // 0 -1 range
    *rho1= hiprand_uniform(&state);
    *rho2= hiprand_uniform(&state);
    *rho3= hiprand_uniform(&state);
}


__global__ void initial_position_velocity(unsigned seed, struct Particle *particles)
{

    int i= blockIdx.x*blockDim.x + threadIdx.x;

	if(i<P_NUM){

    double rho1, rho2, rho3;
    double miu;

    hiprandState state;
    hiprand_init(seed, i, 0, &state);

    bool planet;  // true: Earth ; false: Moon

    if (hiprand_uniform(&state)>0.5)
        particles[i].p_type = true  ;  // silicate particle
    else
        particles[i].p_type = false ;  // iron particle

    if (hiprand_uniform(&state)>0.5)
        planet = true   ;  // silicate particle
    else
        planet = false ;  // iron particle
    
    
    if (planet == true) {   
        if (particles[i].p_type == true) {
            // position initialization for outer shell, silicate particles
            generate_uniform_random_number(seed, i, &rho1, &rho2, &rho3 );
            miu= 1- 2 * rho2;   
            particles[i].position.x = cbrt(pow(R1,3.0)+(pow(R2,3.0)-pow(R1,3.0))*rho1) * sqrt(1-pow(miu,2.0))*cos(2*PI*rho3)+ CENTER_MASS_X;
            particles[i].position.y = cbrt(pow(R1,3.0)+(pow(R2,3.0)-pow(R1,3.0))*rho1) * sqrt(1-pow(miu,2.0))*sin(2*PI*rho3)+ CENTER_MASS_Z;
            particles[i].position.z = cbrt(pow(R1,3.0)+(pow(R2,3.0)-pow(R1,3.0))*rho1) * miu;
        }
        else {
            // position initialization for inner core, iron particles   
            generate_uniform_random_number(seed, i, &rho1, &rho2, &rho3 );
            miu= 1- 2 * rho2;
            particles[i].position.x = R * cbrt(rho1) * sqrt(1-pow(miu,2.0)) * cos(2*PI*rho3) + CENTER_MASS_X;
            particles[i].position.y = R * cbrt(rho1) * sqrt(1-pow(miu,2.0)) * sin(2*PI*rho3) + CENTER_MASS_Z;
            particles[i].position.z = R * cbrt(rho1) * miu;
        }

        // velocity initialization
        particles[i].velocity.x = Init_V;
        particles[i].velocity.y = 0;
        particles[i].velocity.z = 0;
        // calculate the distance r_xz on the plane xz from the center of mass for INER
        float r_xz = sqrt(pow((particles[i].position.x + CENTER_MASS_X),2.0) + pow((particles[i].position.z + CENTER_MASS_Z),2.0));
        float theta = atan((particles[i].position.z + CENTER_MASS_Z) / (particles[i].position.x + CENTER_MASS_X));
        particles[i].velocity.x +=  OMEGA * r_xz * sin(theta);
        particles[i].velocity.z +=  -OMEGA * r_xz * cos(theta);
        particles[i].velocity.y +=  0;
    }
    else {
        if (particles[i].p_type == true) {
            // position initialization for outer shell, silicate particles
            generate_uniform_random_number(seed, i, &rho1, &rho2, &rho3 );
            miu= 1- 2 * rho2;   
            particles[i].position.x = cbrt(pow(R1,3.0)+(pow(R2,3.0)-pow(R1,3.0))*rho1) * sqrt(1-pow(miu,2.0))*cos(2*PI*rho3)- CENTER_MASS_X;
            particles[i].position.y = cbrt(pow(R1,3.0)+(pow(R2,3.0)-pow(R1,3.0))*rho1) * sqrt(1-pow(miu,2.0))*sin(2*PI*rho3)- CENTER_MASS_Z;
            particles[i].position.z = cbrt(pow(R1,3.0)+(pow(R2,3.0)-pow(R1,3.0))*rho1) * miu;
        }
        else {
            // position initialization for inner core, iron particles   
            generate_uniform_random_number(seed, i, &rho1, &rho2, &rho3 );
            miu= 1- 2 * rho2;
            particles[i].position.x = R * cbrt(rho1) * sqrt(1-pow(miu,2.0)) * cos(2*PI*rho3) - CENTER_MASS_X;
            particles[i].position.y = R * cbrt(rho1) * sqrt(1-pow(miu,2.0)) * sin(2*PI*rho3) - CENTER_MASS_Z;
            particles[i].position.z = R * cbrt(rho1) * miu;
        }

        // velocity initialization
        particles[i].velocity.x = -1*Init_V;
        particles[i].velocity.y = 0;
        particles[i].velocity.z = 0;
        // calculate the distance r_xz on the plane xz from the center of mass for INER
        float r_xz = sqrt(pow((particles[i].position.x - CENTER_MASS_X),2.0) + pow((particles[i].position.z - CENTER_MASS_Z),2.0));
        float theta = atan((particles[i].position.z - CENTER_MASS_Z) / (particles[i].position.x - CENTER_MASS_X));
        particles[i].velocity.x +=  OMEGA * r_xz * sin(theta);
        particles[i].velocity.z +=  -OMEGA * r_xz * cos(theta);
        particles[i].velocity.y +=  0;
    }
}
    
}

// interaction forces

__device__ double interactionForce(double radius, double sqr_r, int force_type, bool if_sep_dec){
// radius is the parapmeter 'r' in Table 2
    double M1, M2, K1, K2, KRP1, KRP2;
    double accel;
    if (force_type == 0) {  // accel for silicate particle, silicate-iron 
        M1 = Mfe;
		M2 = Msi;
        K1 = Ksi;
        K2 = Kfe;
        KRP1 = KRPsi;
        KRP2 = KRPfe;
    }
    else if (force_type == 1) { // accel for iron particle, iron-silicate
        M1 = Msi;
		M2 = Mfe;
        K1 = Ksi;
        K2 = Kfe;
        KRP1 = KRPsi;
        KRP2 = KRPfe;
    }
    else if (force_type == 2) { // accel for silicate particle, silicate-silicate
        M1 = Msi;
		M2 = Msi;
        K1 = Ksi;
        K2 = Ksi;
        KRP1 = KRPsi;
        KRP2 = KRPsi;
    }
    else if (force_type == 3) { // accel for iron particle, iron-iron
        M1 = Mfe;
		M2 = Mfe;
        K1 = Kfe;
        K2 = Kfe;
        KRP1 = KRPfe;
        KRP2 = KRPfe;
    }
    else {
      //  fprintf(stderr,"invalid force type");
    }


    if (radius >= D){
        accel = G * M1 / sqr_r;
    }  
    else if (radius >= D-D*SDPsi){
        accel = G * M1 / sqr_r - 0.5*(K1+K2)*(D*D-sqr_r)/M2;
    }

    else if (radius >= D-D*SDPfe && if_sep_dec==true){
        accel = G * M1 / sqr_r - 0.5*(K1+K2)*(D*D-sqr_r)/M2 ;
    }
    else if (radius >= D-D*SDPfe && if_sep_dec==false){
        accel = G * M1 / sqr_r - 0.5*(K1*KRP1+K2)*(D*D-sqr_r)/M2 ;
    }
    else if (radius >= Epsilon  && if_sep_dec==true){
        accel = G * M1 / sqr_r - 0.5*(K1+K2)*(D*D-sqr_r)/M2 ;
    }
    else if (radius >= Epsilon  && if_sep_dec==false){
        accel = G * M1 / sqr_r - 0.5*(K1*KRP1+K2*KRP2)*(D*D-sqr_r)/M2 ;
    }
    else if(radius < Epsilon && radius > 0.11){
//        radius=Epsilon;
		sqr_r = Epsilon*Epsilon;
        accel = G * M1 / sqr_r - 0.5*(K1+K2)*(D*D-sqr_r)/M2 ;
    }
	else {
//		printf("same particles detected!\n");
		accel = 0;
	}
//	if ((accel > 10) || (accel < -10))
//		printf("accel is : %f; radius is : %f; sqr_r is %f\n", accel, radius, sqr_r);

    return accel;
}


__device__ struct vecfloat3 bodyBodyInteraction(struct Particle pi, struct Particle pj, struct vecfloat3 acc)
{
  struct vecfloat3 r;
  struct vecfloat3 v;
  struct vecfloat3 r_next;
  double accel;
  // r_ij  [3 FLOPS]
  r.x = pj.position.x - pi.position.x;
  r.y = pj.position.y - pi.position.y;
  r.z = pj.position.z - pi.position.z;
//  printf("r is: %f, %f, %f\n", r.x, r.y, r.z);

  v.x = pj.velocity.x - pi.velocity.x;
  v.y = pj.velocity.y - pi.velocity.y;
  v.z = pj.velocity.z - pi.velocity.z;
//  printf("v is: %f, %f, %f\n", v.x, v.y, v.z);

  r_next.x = r.x + v.x * Time_step;
  r_next.y = r.y + v.y * Time_step;
  r_next.z = r.z + v.z * Time_step;

  int force_type;
  bool if_sep_dec;
  float sqr_r = r.x * r.x + r.y * r.y + r.z * r.z;
  float sqr_r_next = r_next.x * r_next.x + r_next.y * r_next.y + r_next.z * r_next.z;
  if (sqr_r_next < sqr_r) {
    if_sep_dec = 1;
  }
  sqr_r += softeningSquared;
  float radius = sqrtf(sqr_r);

  if ( pi.p_type && !pj.p_type ) {
    force_type = 0;
  } 
  else if ( !pi.p_type && pj.p_type ) {
    force_type = 1;
  }
  else if ( pi.p_type && pj.p_type ) {
    force_type = 2;
  }
  else if ( !pi.p_type && !pj.p_type ) {
    force_type = 3;
  }
  else {
    // fprintf(stderr,"invalid force type!");
  }
  accel = interactionForce(radius, sqr_r, force_type, if_sep_dec);
  
  acc.x += r.x / radius * accel;
  acc.y += r.y / radius * accel;
  acc.z += r.z / radius * accel;

//  double abs_acc = acc.x*acc.x + acc.y*acc.y + acc.z*acc.z;
//  if ((abs_acc > 100) || (abs_acc < -100)){
//		printf("accel is: %f; pj is: %f, %f, %f; pi is: %f, %f, %f\n", accel, pj.position.x, pj.position.y, pj.position.z, pi.position.x, pi.position.y, pi.position.z);
//  }
  return acc;
}

__device__ struct vecfloat3 tile_calculation(struct Particle myParticle, struct vecfloat3 acc)
{
  int i;
  extern __shared__ struct Particle shParticles[];
  for (i = 0; i < blockDim.x; i++) {
	if (isnan(myParticle.position.x) || isnan(shParticles[i].position.x)) 
		printf("i is %d\n", i);
	else
    	acc = bodyBodyInteraction(myParticle, shParticles[i], acc);
  }
  return acc;
}

__global__ void calculate_forces(struct Particle *devP, struct vecfloat3 *devA)
{
  extern __shared__ struct Particle shParticles[]; 
  struct Particle myParticle;
  int i, tile;
  struct vecfloat3 acc; //= {0.0f, 0.0f, 0.0f}
  acc.x = 0.0f;
  acc.y = 0.0f;
  acc.z = 0.0f;
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gtid < P_NUM){
    myParticle = devP[gtid];
	if (isnan(myParticle.position.x)) 
		printf("gtid is %d\n", gtid);
  	for (i = 0, tile = 0; i < P_NUM; i += TPB, tile++) {
    	int idx = tile * blockDim.x + threadIdx.x;
    	shParticles[threadIdx.x] = devP[idx];
    	__syncthreads();
    	acc = tile_calculation(myParticle, acc);
    	__syncthreads();
  	}
	devA[gtid] = acc;
//	double abs_acc = acc.x*acc.x + acc.y*acc.y + acc.z*acc.z;
//	if (abs_acc > 1000)
//		printf("gtid is: %d; acc is: %.12f, %.12f, %.12f\n", gtid, acc.x, acc.y, acc.z);

  } 

}

__global__ void update_pos(struct Particle *devP, struct vecfloat3 *devA)
{

  int i= blockIdx.x*blockDim.x + threadIdx.x;

  if(i<P_NUM){
  	devP[i].position.x = devP[i].position.x + devP[i].velocity.x * Time_step + devA[i].x / 2.0f * pow(Time_step, 2.0); 
  	devP[i].position.y = devP[i].position.y + devP[i].velocity.y * Time_step + devA[i].y / 2.0f * pow(Time_step, 2.0); 
  	devP[i].position.z = devP[i].position.z + devP[i].velocity.z * Time_step + devA[i].z / 2.0f * pow(Time_step, 2.0); 
	double abs_devA = devA[i].x*devA[i].x + devA[i].y*devA[i].y + devA[i].z*devA[i].z;
//	if (abs_devA > 100)
//		printf("devA in update_vel is: %f, %f, %f\n", devA[i].x, devA[i].y, devA[i].z);
//  }
	
}

__global__ void update_vel(struct Particle *devP, struct vecfloat3 *devA, struct vecfloat3 *devA_next)
{

  int i= blockIdx.x*blockDim.x + threadIdx.x;

  if(i<P_NUM){
  	devP[i].velocity.x = devP[i].velocity.x + (devA_next[i].x + devA[i].x) / 2.0f * Time_step; 
  	devP[i].velocity.y = devP[i].velocity.y + (devA_next[i].y + devA[i].y) / 2.0f * Time_step; 
  	devP[i].velocity.z = devP[i].velocity.z + (devA_next[i].z + devA[i].z) / 2.0f * Time_step; 
  }


}

__global__ void print_devP(struct Particle *devP, struct vecfloat3 *devA)
{

  int i= blockIdx.x*blockDim.x + threadIdx.x;
  if(i<P_NUM)
	printf("GPU particle position is: %f, %f, %f\n", devP[i].position.x, devP[i].position.y, devP[i].position.z);
}


void particle_init(unsigned seed, struct Particle *cpuP)
{
	struct Particle *devP=0;

	hipMalloc(&devP, P_NUM*sizeof(struct Particle));
	
    initial_position_velocity<<<(P_NUM+TPB-1)/TPB, TPB>>>(seed, devP);

    hipDeviceSynchronize();

    hipMemcpy(cpuP, devP, P_NUM*sizeof(struct Particle),hipMemcpyDeviceToHost);
	hipFree(devP);
}


void particle_update(struct Particle *cpuP)
{
	struct Particle *devP=0;
	struct vecfloat3 *devA=0;
	struct vecfloat3 *devA_next=0;
	checkCudaErrors(hipMalloc(&devP, P_NUM*sizeof(struct Particle)));
	checkCudaErrors(hipMalloc(&devA, P_NUM*sizeof(struct vecfloat3)));
	checkCudaErrors(hipMalloc(&devA_next, P_NUM*sizeof(struct vecfloat3)));
 
	checkCudaErrors(hipMemcpy(devP, cpuP, P_NUM*sizeof(struct Particle),hipMemcpyHostToDevice));
	hipDeviceSynchronize();

//	printf("calc force 1 cpuP position is: %f, %f, %f\n", cpuP[10].position.x, cpuP[10].position.y, cpuP[10].position.z);
    // update position
    calculate_forces<<<(P_NUM+TPB-1)/TPB, TPB, TPB*sizeof(struct Particle)>>>(devP, devA);
    getLastCudaError("Kernel execution failed");  	// check if kernel execution generated and error
    hipDeviceSynchronize();
//	checkCudaErrors(hipMemcpy(cpuP, devP, P_NUM*sizeof(struct Particle),hipMemcpyDeviceToHost));
//	printf("calc force 2 cpuP position is: %f, %f, %f\n", cpuP[10].position.x, cpuP[10].position.y, cpuP[10].position.z);

    update_pos<<<(P_NUM+TPB-1)/TPB, TPB>>>(devP, devA);
	getLastCudaError("Kernel execution failed");  	// check if kernel execution generated and error
    hipDeviceSynchronize();
//	checkCudaErrors(hipMemcpy(cpuP, devP, P_NUM*sizeof(struct Particle),hipMemcpyDeviceToHost));
//	printf("calc force 3 cpuP position is: %f, %f, %f\n", cpuP[10].position.x, cpuP[10].position.y, cpuP[10].position.z);
	

    // update velocity
    calculate_forces<<<(P_NUM+TPB-1)/TPB, TPB, TPB*sizeof(struct Particle)>>>(devP, devA_next);
	getLastCudaError("Kernel execution failed");  	// check if kernel execution generated and error
    hipDeviceSynchronize();
//	checkCudaErrors(hipMemcpy(cpuP, devP, P_NUM*sizeof(struct Particle),hipMemcpyDeviceToHost));
//	printf("calc force 3 cpuP position is: %f, %f, %f\n", cpuP[10].position.x, cpuP[10].position.y, cpuP[10].position.z);
    update_vel<<<(P_NUM+TPB-1)/TPB, TPB>>>(devP, devA, devA_next);
	getLastCudaError("Kernel execution failed");  	// check if kernel execution generated and error
    hipDeviceSynchronize();
	
	checkCudaErrors(hipMemcpy(cpuP, devP, P_NUM*sizeof(struct Particle),hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(devP));
    checkCudaErrors(hipFree(devA));
    checkCudaErrors(hipFree(devA_next));

}

//void cuda_release()
//{
//    hipFree(devP);
//    hipFree(devA);
//    hipFree(devA_next);
//}




